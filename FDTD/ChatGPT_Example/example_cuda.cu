#include "hip/hip_runtime.h"
// gpu_ops.cu
#include "gpu_ops.h"
#include "utils.h"
#include <hip/hip_runtime.h>

// CUDA kernel that performs OR reduction for double values
__global__ void any_greater_than_kernel(const double* d_data, bool* d_block_results, int N, double threshold) {
    __shared__ bool shared[256];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    bool gt = false;
    if (idx < N) {
        gt = d_data[idx] > threshold;
    }

    shared[tid] = gt;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared[tid] = shared[tid] || shared[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
        d_block_results[blockIdx.x] = shared[0];
}

// Host function that launches the kernel and returns the final result
bool anyValueGreaterThan(const double* h_data, int size, double threshold) {
    const int blockSize = 256;
    const int numBlocks = (size + blockSize - 1) / blockSize;

    // Allocate device memory
    double* d_data = nullptr;
    checkCuda(hipMalloc(&d_data, size * sizeof(double)), "hipMalloc d_data");
    checkCuda(hipMemcpy(d_data, h_data, size * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy d_data");

    // Allocate memory for block-level results
    bool* d_block_results = nullptr;
    checkCuda(hipMalloc(&d_block_results, numBlocks * sizeof(bool)), "hipMalloc d_block_results");

    // Launch kernel
    any_greater_than_kernel<<<numBlocks, blockSize>>>(d_data, d_block_results, size, threshold);
    checkCuda(hipGetLastError(), "Kernel launch failed");
    checkCuda(hipDeviceSynchronize(), "Kernel execution failed");

    // Copy back block results
    std::vector<bool> h_block_results(numBlocks);
    checkCuda(hipMemcpy(h_block_results.data(), d_block_results, numBlocks * sizeof(bool), hipMemcpyDeviceToHost), "Memcpy block results");

    // Final reduction on host
    bool final_result = false;
    for (bool r : h_block_results) {
        final_result = final_result || r;
    }

    // Free memory
    hipFree(d_data);
    hipFree(d_block_results);

    return final_result;
}
