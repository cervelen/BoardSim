#include "hip/hip_runtime.h"
#include "fdtd_experiment.h"
//#include <hip/hip_runtime.h>



D bool sphere::in_sphere(double3 point){
    return true;
}

__global__ void check_in_sphere(sphere my_sphere, double3* points, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        my_sphere.in_sphere(points[idx]);
    }
}

void launch_check_in_sphere(sphere my_sphere, double3* points, int n){
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    check_in_sphere<<<gridSize, blockSize>>>(my_sphere, points, n);
    hipDeviceSynchronize();
}