#include "hip/hip_runtime.h"
#include "fdtd.h"
#include <hip/hip_runtime.h>

struct sphere{
    
}
__global__ void divide_kernel(double* numerator, double* denominator, double* result, int size){ //each vector is the same size
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size){
        if (denominator[idx] != 0.0) {
            result[idx] = numerator[idx] / denominator[idx];
        } else {
            result[idx] = 0.0;
        }
    }
}
void divide_vectors(double* numerator, double* denominator, double* result, int size){

    //allocates memory on the CUDA side
    double *d_numerator, *d_denominator, *d_result;
    hipMalloc(&d_numerator, size * sizeof(double));
    hipMalloc(&d_denominator, size * sizeof(double));
    hipMalloc(&d_result, size * sizeof(double));

    //copies values from host to cuda device
    hipMemcpy(d_numerator, numerator, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_denominator, denominator, size * sizeof(double), hipMemcpyHostToDevice);

    //generates number of blocks needed for division of all doubles
    int threadsPerBlock = 128;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    //runs divide_kernel
    divideKernel<<<numBlocks, threadsPerBlock>>>(d_numerator, d_denominator, d_result, size);

    //copies memory back to host
    hipMemcpy(result, d_result, size * sizeof(double), hipMemcpyDeviceToHost);
    
    //cleans up cuda memory
    hipFree(d_numerator);
    hipFree(d_denominator);
    hipFree(d_result);


}