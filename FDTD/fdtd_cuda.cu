int main() {
    const int N = 10000;              // Size of input array
    double threshold = 100.0;         // Value to compare against

    // Allocate and initialize host array
    double* h_data = new double[N];
    for (int i = 0; i < N; ++i)
        h_data[i] = static_cast<double>(i) / 10.0;  // e.g. 0.0, 0.1, ..., 999.9

    // Optional: to test the "false" case, uncomment this line
    // h_data[N - 1] = 99.9;  // Ensures all elements are below the threshold

    // Allocate device memory for input array
    double* d_data;
    checkCuda(hipMalloc(&d_data, N * sizeof(double)), "hipMalloc d_data");

    // Copy data from host to device
    checkCuda(hipMemcpy(d_data, h_data, N * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy d_data");

    // Configure kernel launch parameters
    const int blockSize = 256;  // Number of threads per block
    const int numBlocks = (N + blockSize - 1) / blockSize;  // Total number of blocks needed

    // Allocate memory to store one bool result per block
    bool* d_block_results;
    checkCuda(hipMalloc(&d_block_results, numBlocks * sizeof(bool)), "hipMalloc d_block_results");

    // Launch the CUDA kernel to compare and reduce within each block
    any_double_greater_than<<<numBlocks, blockSize>>>(
        d_data, d_block_results, N, threshold
    );
    checkCuda(hipGetLastError(), "Kernel launch failed");

    // Allocate host memory to retrieve block-level results
    bool* h_block_results = new bool[numBlocks];

    // Copy results from device to host
    checkCuda(hipMemcpy(h_block_results, d_block_results, numBlocks * sizeof(bool), hipMemcpyDeviceToHost), "Memcpy block results");

    // Final OR reduction on the host across all block results
    bool final_result = false;
    for (int i = 0; i < numBlocks; ++i)
        final_result = final_result || h_block_results[i];

    // Print the final result
    std::cout << "Any value greater than " << threshold << "? " << std::boolalpha << final_result << std::endl;

    // Cleanup device and host memory
    hipFree(d_data);
    hipFree(d_block_results);
    delete[] h_data;
    delete[] h_block_results;

    return 0;
}
